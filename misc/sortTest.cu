#include "hip/hip_runtime.h"
#include <cstdio>
#include <cstdlib>
#include <cassert>
#include <vector>
#include <algorithm>
//#include <sort>
#include "../treebuild/rtc.h"
#include "../treebuild/cudamem.h"

//Compile: nvcc -O3 -o sortTest  sortTest.cu -arch=sm_30 -Xptxas=-v

#define IDSHIFT 24
#define VALMASK 0x0000000F

#define WARP_SIZE2 5
#define WARP_SIZE 32

// -arch=sm_35 -Xptxas=-v -lcudadevrt -rdc=true -g  -DPLUMMER -DNPERLEAF=16 -maxrregcount=32

struct cmp_3bits{
  bool operator () (const uint &a, const uint &b){
    //Strip the ID
    int anew = a & VALMASK;
    int bnew = b & VALMASK;
    return anew < bnew;
  }
};

int hostSortTest(uint *in_data, uint *out_data, uint n)
{
  //Add the index to the data
  for(int i=0; i < n; i++)
  {
    out_data[i] = (i << IDSHIFT) | in_data[i];
  }

  //sort
  std::stable_sort(out_data, out_data+n, cmp_3bits());


  return 0;
}

void kernelSuccess(const char kernel[] = "kernel")
{
  const int ret = (hipDeviceSynchronize() != hipSuccess);
  if (ret)
  {
    fprintf(stderr, "%s launch failed: %s\n", kernel, hipGetErrorString(hipGetLastError()));
    assert(0);
  }
}

//#define PRINT_STATS

//Basic bitonic sort, taken from the Advanced Quicksort example in the SDK as reference implementation
static __global__ void testSortKernel_bitonicSDK(const int n, uint *input, uint *output)
{
  const int tid = threadIdx.x;
  const int bid = blockIdx.x;
  const int idx = bid*blockDim.x + tid;

  #ifdef PRINT_STATS
    int loopCount = 0;
    int swaps     = 0;
  #endif

  __shared__ int sortbuf[256];

  sortbuf[tid] = (tid << IDSHIFT) | input[idx];
  __syncthreads();

//   bitonicsort_kernel(input, output, 0, n);
  //Call sort rountines
 // Now the sort loops
  // Here, "k" is the sort level (remember bitonic does a multi-level butterfly style sort)
  // and "j" is the partner element in the butterfly.
  // Two threads each work on one butterfly, because the read/write needs to happen
  // simultaneously
  for(unsigned int k=2; k<=blockDim.x; k*=2)  // Butterfly stride increments in powers of 2
  {
    for(unsigned int j=k>>1; j>0; j>>=1) // Strides also in powers of to, up to <k
    {
      unsigned int swap_idx = threadIdx.x ^ j; // Index of element we're compare-and-swapping with
      unsigned my_elem      = sortbuf[threadIdx.x];
      unsigned swap_elem    = sortbuf[swap_idx];

      __syncthreads();

      #ifdef PRINT_STATS
        loopCount++;
      #endif

      // The k'th bit of my threadid (and hence my sort item ID)
      // determines if we sort ascending or descending.
      // However, since threads are reading from the top AND the bottom of
      // the butterfly, if my ID is > swap_idx, then ascending means mine<swap.
      // Finally, if either my_elem or swap_elem is out of range, then it
      // ALWAYS acts like it's the largest number.
      // Confusing? It saves us two writes though.
      unsigned int ascend  = k * (swap_idx < threadIdx.x);
      unsigned int descend = k * (swap_idx > threadIdx.x);
      bool swap = false;
      if((threadIdx.x & k) == ascend)
        {
          if((my_elem & VALMASK) > (swap_elem & VALMASK))
            swap = true;
        }
        if((threadIdx.x & k) == descend)
        {
          if((my_elem & VALMASK) < (swap_elem & VALMASK))
            swap = true;
        }

        // If we had to swap, then write my data to the other element's position.
        // Don't forget to track out-of-range status too!
        if(swap)
        {
          sortbuf[swap_idx] = my_elem;
          #ifdef PRINT_STATS
            swaps++;
          #endif
        }

        __syncthreads();
      }
    }


    #ifdef PRINT_STATS
      printf("[%d, %d ]\t Loops: %d Swaps: %d \n", bid, tid, loopCount, swaps);
    #endif

  //Combine the value and the thread-id into the results

   output[idx] = sortbuf[tid];
}

//Custom sort method
static __device__ __forceinline__ int lanemask_lt()
{
  int mask;
  asm("mov.u32 %0, %lanemask_lt;" : "=r" (mask));
  return mask;
}
static __device__ __forceinline__ int2 warpBinExclusiveScan(const bool p)
{
  const unsigned int b = __ballot(p);
  return make_int2(__popc(b & lanemask_lt()), __popc(b));
}

// static __device__ __forceinline__ int warpBinReduce(const bool p)
// {
//   const unsigned int b = __ballot(p);
//   return __popc(b);
// }

static __device__ __forceinline__ uint shfl_scan_add_step(const uint partial, const uint up_offset)
{
  uint result;
  asm(
      "{.reg .u32 r0;"
      ".reg .pred p;"
      "shfl.up.b32 r0|p, %1, %2, 0;"
      "@p add.u32 r0, r0, %3;"
      "mov.u32 %0, r0;}"
      : "=r"(result) : "r"(partial), "r"(up_offset), "r"(partial));
  return result;
}


static __global__ void testSortKernel(const int n, uint *input, uint *output)
{
  const int tid = threadIdx.x;
  const int bid = blockIdx.x;
  const int idx = bid*blockDim.x + tid;

  const int laneIdx = threadIdx.x & (WARP_SIZE-1);
  const int warpIdx = threadIdx.x >> WARP_SIZE2;

  __shared__ int sortbuf[65]; //2*32 + 1, +1 for value exchange

  //Put the to be sorted values into shared memory
  uint value = (tid << IDSHIFT) | input[idx];
  __syncthreads();

  //Histogram count, first each warp makes a local histogram

  //int2 histogram[2]; //x will contain the offset within the warp
  int2 histogram;
                     //y will contain the total sum of the warp
                     //Index 0, is not used only to prevent if
                     //Index 1, contains the result for the threads value

  const int val = (value & 0x0000000F);
  int2 scanRes;
  //Count per radix the offset and number of values
  #pragma unroll
  for(int i=0; i < 8; i++)
  {
    scanRes = warpBinExclusiveScan((val == i));

    if(laneIdx == i) //Lane 0 to 8, directly write the scan sum to shared-mem
    {
      sortbuf[laneIdx*8+warpIdx] = scanRes.y;
    }
    if(val == i)
      histogram = scanRes; //Index 1 contains the correct value
  }
  __syncthreads();
  //Now compute the prefix sums across our warps
  //note that we have 8 values by 8 histogram values
  //store this 8x8 into 64 lanes.
  //warp0_hist0, warp1_hist0, warp2_hist0, ...warp6_hist7, warp7_hist7
  //this allows us to compute the prefix sum using two warps


  //Compute the exclusive prefix sum, using binary reduction
  //            0, 1, 2, 3, 4, 5, 6, 7
  //            0 + 1, 2+3, 4+5, 6+7
  //              A  + B  ,  C  + D
  //                 E    +    F
  int offset;
  if(warpIdx < 2)
  {
    offset =  sortbuf[laneIdx+WARP_SIZE*warpIdx];
    #pragma unroll
      for(int i = 0; i < 5; i++) /* log2(32) steps */
        offset = shfl_scan_add_step(offset, 1 << i);

    //Now we have two warps with prefix sums, we need to add the final value
    //of the first warp to the values of the second warp. Use the unused location
    if(threadIdx.x==31) sortbuf[64] = offset;

    offset -= sortbuf[laneIdx+WARP_SIZE*warpIdx]; //Make exclusive
  }
  __syncthreads(); //Wait on sortbuf[64] to be stored

  if(warpIdx == 1)
  {
    offset+=sortbuf[64];
  }
  
  //Prefix sum is done, write out the results
  if(warpIdx < 2)
  {
    sortbuf[laneIdx+WARP_SIZE*warpIdx] = offset;
  }
  __syncthreads();
  
  //Now each thread reads their storage location in the following way:
  //Value to read is one of the eight bins, namely the one associated to
  //the value and also is offset by the warp
  
   //val*8 + warpIdx
   //int storeLocation = sortbuf[val*8 + warpIdx] + histogram[1].x; //per warp offset+in-warp offset
   int storeLocation = sortbuf[val*8 + warpIdx] + histogram.x; //per warp offset+in-warp offset

   //Coalesced output
   output[bid*blockDim.x+storeLocation] = value;
}



int main(int argc, char * argv [])
{
  const int nPerThread = 256;
  int nBlocks          = 1024;
  

  if(argc > 1)
    nBlocks = atoi(argv[1]);

  const int n = nPerThread*nBlocks;

  host_mem<uint> h_input, h_output, h_check;
  h_input.alloc(n);
  h_output.alloc(n);
  h_check.alloc(n);

  cuda_mem<uint> d_input, d_output;
  d_input.alloc(n);
  d_output.alloc(n);

  int histoCount[8] = {0};
  //Allocate some data
  for(int i=0; i < n; i++)
  {
    h_input[i] = ((int)(1000*drand48())) % 8;
    //fprintf(stderr, "%d\t->\t%d\n", i, h_input[i]);
    histoCount[h_input[i]]++;
  }

  for(int i=0; i < 8; i++)
	  fprintf(stdout,"Data-stats: %d\t%d\n",
			  i, histoCount[i]);

  d_input.h2d(h_input);

  //Call the sort kernel
  const int NBLOCKS  = nBlocks;
  const int NTHREADS = nPerThread; //Should be 256!!

  double t0 = rtc();
  testSortKernel_bitonicSDK<<<NBLOCKS,NTHREADS>>>(n, d_input, d_output);
  kernelSuccess("testSortKernel_bitonicSDK");
  double t1 = rtc();
  testSortKernel<<<NBLOCKS,NTHREADS>>>(n, d_input, d_output);
  kernelSuccess("testSortKernel");
  double t2 = rtc();
  d_output.d2h(h_output);


  //Compute result on the host
  double t3 = rtc();
  for(int i=0; i < n; i+=256)
  {
    hostSortTest(h_input+i, h_check+i, 256);
  }
  double t4 = rtc();

  const int printStride = 1;
  int matchCount = 0;
  int matchValCount = 0;
  int matchIDCount  = 0;
  for(int i=0; i < n; i+=printStride)
  {
    if(i < n)
    {
      //Extract id and value
      uint id  = h_output[i] >> IDSHIFT;
      int val  = h_output[i] & VALMASK;

      uint hid  = h_check[i] >> IDSHIFT;
      int hval  = h_check[i] & VALMASK;

      int match_id = 0;
      int match_val= 0;
      if(id == hid)
        match_id = 1;
      if(val == hval)
        match_val = 1;
    
      matchValCount += match_val;
      matchIDCount  += match_id;
      if(match_id && match_val) matchCount++;

   /*
      if(match_id == 0 || match_val == 0)
        fprintf(stderr, "Index: %d Error GPU: (%d, %d)\tCPU: (%d, %d)  Match-ID: %d  Match-val: %d \n",
                        i, val, id, hval, hid, match_id, match_val);
  */
    }
  }

  fprintf(stdout,"Total items: %d  Match-full: %d Match-val: %d Match-id: %d \n", 
                  n, matchCount, matchValCount, matchIDCount);
  fprintf(stdout,"Time host: %lg  Time bitonic: %lg   Time-radix: %lg \n", 
                  t4-t3, t1-t0, t2-t1);
  fprintf(stdout,"Time-radix: %lg %f MPtcl/s\n", 
                  t2-t1,  ((1/(t2-t1))*n)/1000000);

}
