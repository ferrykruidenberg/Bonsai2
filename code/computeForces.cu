#include "hip/hip_runtime.h"
#include "Treecode.h"

#if 1
namespace computeForces
{
  static __device__ __forceinline__ int lanemask_lt()
  {
    int mask;
    asm("mov.u32 %0, %lanemask_lt;" : "=r" (mask));
    return mask;
  }
  static __device__ __forceinline__ uint shfl_scan_add_step(uint partial, uint up_offset)
  {
    uint result;
    asm(
        "{.reg .u32 r0;"
        ".reg .pred p;"
        "shfl.up.b32 r0|p, %1, %2, 0;"
        "@p add.u32 r0, r0, %3;"
        "mov.u32 %0, r0;}"
        : "=r"(result) : "r"(partial), "r"(up_offset), "r"(partial));
    return result;
  }
  template <const int levels>
    static __device__ __forceinline__ uint inclusive_scan_warp(const int sum)
    {
      uint mysum = sum;
#pragma unroll
      for(int i = 0; i < levels; ++i)
        mysum = shfl_scan_add_step(mysum, 1 << i);
      return mysum;
    }

  static __device__ __forceinline__ int2 warpIntExclusiveScan(const int value)
  {
    const int sum = inclusive_scan_warp<WARP_SIZE2>(value);
    return make_int2(sum-value, __shfl(sum, WARP_SIZE-1, WARP_SIZE));
  }

  static __device__ __forceinline__ int2 warpBinExclusiveScan(const bool p)
  {
    const unsigned int b = __ballot(p);
    return make_int2(__popc(b & lanemask_lt()), __popc(b));
  }

  /******************* segscan *******/

  static __device__ __forceinline__ int lanemask_le()
  {
    int mask;
    asm("mov.u32 %0, %lanemask_le;" : "=r" (mask));
    return mask;
  }
  static __device__ __forceinline__ int ShflSegScanStepB(
      int partial,
      uint distance,
      uint up_offset)
  {
    asm(
        "{.reg .u32 r0;"
        ".reg .pred p;"
        "shfl.up.b32 r0, %1, %2, 0;"
        "setp.le.u32 p, %2, %3;"
        "@p add.u32 %1, r0, %1;"
        "mov.u32 %0, %1;}"
        : "=r"(partial) : "r"(partial), "r"(up_offset), "r"(distance));
    return partial;
  }
  template<const int SIZE2>
    static __device__ __forceinline__ int inclusive_segscan_warp_step(int value, const int distance)
    {
      for (int i = 0; i < SIZE2; i++)
        value = ShflSegScanStepB(value, distance, 1<<i);
      return value;
    }
  static __device__ __forceinline__ int2 inclusive_segscan_warp(
      const int packed_value, const int carryValue)
  {
    const int  flag = packed_value < 0;
    const int  mask = -flag;
    const int value = (~mask & packed_value) + (mask & (-1-packed_value));

    const int flags = __ballot(flag);

    const int dist_block = __clz(__brev(flags));

    const int laneIdx = threadIdx.x & (WARP_SIZE - 1);
    const int distance = __clz(flags & lanemask_le()) + laneIdx - 31;
    const int val = inclusive_segscan_warp_step<WARP_SIZE2>(value, min(distance, laneIdx));
    return make_int2(val + (carryValue & (-(laneIdx < dist_block))), __shfl(val, WARP_SIZE-1, WARP_SIZE));
  }



#define NCRIT 64
#define CELL_LIST_MEM_PER_WARP (2048*32)

  template<int SHIFT>
    __forceinline__ static __device__ int ringAddr(const int i)
    {
      return (i & ((CELL_LIST_MEM_PER_WARP<<SHIFT) - 1));
    }

  texture<uint4,  1, hipReadModeElementType> texCellData;
  texture<float4, 1, hipReadModeElementType> texCellSize;
  texture<float4, 1, hipReadModeElementType> texCellMonopole;
  texture<float4, 1, hipReadModeElementType> texCellQuad0;
  texture<float4, 1, hipReadModeElementType> texCellQuad1;
  texture<float4, 1, hipReadModeElementType> texPtcl;

  /*******************************/
  /****** Opening criterion ******/
  /*******************************/

  //Improved Barnes Hut criterium
  static __device__ bool split_node_grav_impbh(
      const float4 nodeCOM, 
      const float4 groupCenter, 
      const float4 groupSize)
  {
    //Compute the distance between the group and the cell
    float3 dr = make_float3(
        fabsf(groupCenter.x - nodeCOM.x) - (groupSize.x),
        fabsf(groupCenter.y - nodeCOM.y) - (groupSize.y),
        fabsf(groupCenter.z - nodeCOM.z) - (groupSize.z)
        );

    dr.x += fabsf(dr.x); dr.x *= 0.5f;
    dr.y += fabsf(dr.y); dr.y *= 0.5f;
    dr.z += fabsf(dr.z); dr.z *= 0.5f;

    //Distance squared, no need to do sqrt since opening criteria has been squared
    const float ds2    = dr.x*dr.x + dr.y*dr.y + dr.z*dr.z;

    return (ds2 <= fabsf(nodeCOM.w));
  }

  /******* force due to monopoles *********/

  static __device__ __forceinline__ float4 add_acc(
      float4 acc,  const float4 pos,
      const float massj, const float3 posj,
      const float eps2)
  {
    const float3 dr = make_float3(posj.x - pos.x, posj.y - pos.y, posj.z - pos.z);

    const float r2     = dr.x*dr.x + dr.y*dr.y + dr.z*dr.z + eps2;
    const float rinv   = rsqrtf(r2);
    const float rinv2  = rinv*rinv;
    const float mrinv  = massj * rinv;
    const float mrinv3 = mrinv * rinv2;

    acc.w -= mrinv;
    acc.x += mrinv3 * dr.x;
    acc.y += mrinv3 * dr.y;
    acc.z += mrinv3 * dr.z;

    return acc;
  }


  /******* force due to quadrupoles *********/

  static __device__ __forceinline__ float4 add_acc(
      float4 acc, 
      const float4 pos,
      const float mass, const float3 com,
      const float4 Q0,  const float4 Q1, float eps2) 
  {
    const float3 dr = make_float3(pos.x - com.x, pos.y - com.y, pos.z - com.z);
    const float  r2 = dr.x*dr.x + dr.y*dr.y + dr.z*dr.z + eps2;

    const float rinv  = rsqrtf(r2);
    const float rinv2 = rinv *rinv;
    const float mrinv  =  mass*rinv;
    const float mrinv3 = rinv2*mrinv;
    const float mrinv5 = rinv2*mrinv3; 
    const float mrinv7 = rinv2*mrinv5;   // 16

    float  D0  =  mrinv;
    float  D1  = -mrinv3;
    float  D2  =  mrinv5*(  3.0f);
    float  D3  =  mrinv7*(-15.0f); // 3

    const float q11 = Q0.x;
    const float q22 = Q0.y;
    const float q33 = Q0.z;
    const float q12 = Q1.x;
    const float q13 = Q1.y;
    const float q23 = Q1.z;

    const float  q  = q11 + q22 + q33;
    const float3 qR = make_float3(
        q11*dr.x + q12*dr.y + q13*dr.z,
        q12*dr.x + q22*dr.y + q23*dr.z,
        q13*dr.x + q23*dr.y + q33*dr.z);
    const float qRR = qR.x*dr.x + qR.y*dr.y + qR.z*dr.z;  // 22

    acc.w  -= D0 + 0.5f*(D1*q + D2*qRR);
    float C = D1 + 0.5f*(D2*q + D3*qRR);
    acc.x  += C*dr.x + D2*qR.x;
    acc.y  += C*dr.y + D2*qR.y;
    acc.z  += C*dr.z + D2*qR.z;               // 23

    // total: 16 + 3 + 22 + 23 = 64 flops 

    return acc;
  }


  /******* evalue forces from particles *******/
  template<int NI, bool FULL>
    static __device__ __forceinline__ void directAcc(
        float4 acc_i[NI], 
        const float4 pos_i[NI],
        const int ptclIdx,
        const float eps2)
    {
#if 1
      const float4 M0 = (FULL || ptclIdx >= 0) ? tex1Dfetch(texPtcl, ptclIdx) : make_float4(0.0f, 0.0f, 0.0f, 0.0f);

      //#pragma unroll
      for (int j = 0; j < WARP_SIZE; j++)
      {
        const float4 jM0 = make_float4(__shfl(M0.x, j), __shfl(M0.y, j), __shfl(M0.z, j), __shfl(M0.w,j));
        const float  jmass = jM0.w;
        const float3 jpos  = make_float3(jM0.x, jM0.y, jM0.z);
#pragma unroll
        for (int k = 0; k < NI; k++)
          acc_i[k] = add_acc(acc_i[k], pos_i[k], jmass, jpos, eps2);
      }
#endif
    }

  /******* evalue forces from cells *******/
  template<int NI, bool FULL>
    static __device__ __forceinline__ void approxAcc(
        float4 acc_i[NI], 
        const float4 pos_i[NI],
        const int cellIdx,
        const float eps2)
    {
      float4 M0, Q0, Q1;
      if (FULL || cellIdx >= 0)
      {
        M0 = tex1Dfetch(texCellMonopole, cellIdx);
        Q0 = tex1Dfetch(texCellQuad0,    cellIdx);
        Q1 = tex1Dfetch(texCellQuad1,    cellIdx);
      }
      else
        M0 = Q0 = Q1 = make_float4(0.0f, 0.0f, 0.0f, 0.0f);

      for (int j = 0; j < WARP_SIZE; j++)
      {
        const float4 jM0 = make_float4(__shfl(M0.x, j), __shfl(M0.y, j), __shfl(M0.z, j), __shfl(M0.w,j));
        const float4 jQ0 = make_float4(__shfl(Q0.x, j), __shfl(Q0.y, j), __shfl(Q0.z, j), 0.0f);
        const float4 jQ1 = make_float4(__shfl(Q1.x, j), __shfl(Q1.y, j), __shfl(Q1.z, j), 0.0f);
        const float  jmass = jM0.w;
        const float3 jpos  = make_float3(jM0.x, jM0.y, jM0.z);
#pragma unroll
        for (int k = 0; k < NI; k++)
          acc_i[k] = add_acc(acc_i[k], pos_i[k], jmass, jpos, jQ0, jQ1, eps2);
      }
    }



  template<int SHIFT, int BLOCKDIM2, int NI, bool INTCOUNT>
    static __device__ 
    uint2 treewalk(
        float4 acc_i[NI],
        const float4 _pos_i[NI],
        const float4 groupPos,
        const float eps2,
        const uint2 top_cells,
        int *shmem,
        int *cellList,
        const float4 groupSize)
    {
      const int laneIdx = threadIdx.x & (WARP_SIZE-1);

      /* this helps to unload register pressure */
      float4 pos_i[NI];
#pragma unroll 1
      for (int i = 0; i < NI; i++)
        pos_i[i] = _pos_i[i];

      uint2 interactionCounters = {0}; /* # of approximate and exact force evaluations */

      volatile int *tmpList = shmem;

      int approxCellIdx, directPtclIdx;

      int directCounter = 0;
      int approxCounter = 0;

      for (int root_cell = top_cells.x; root_cell < top_cells.y; root_cell += WARP_SIZE)
        if (root_cell + laneIdx < top_cells.y)
          cellList[ringAddr<SHIFT>(root_cell - top_cells.x + laneIdx)] = root_cell + laneIdx;

      int nCells = top_cells.y - top_cells.x;

      int cellListBlock        = 0;
      int nextLevelCellCounter = 0;

      unsigned int cellListOffset = 0;

      /* process level with n_cells */
#if 1
      while (nCells > 0)
      {
        /* extract cell index from the current level cell list */
        const int cellListIdx = cellListBlock + laneIdx;
        const bool useCell    = cellListIdx < nCells;
        const int cellIdx     = cellList[ringAddr<SHIFT>(cellListOffset + cellListIdx)];
        cellListBlock += min(WARP_SIZE, nCells - cellListBlock);

        /* read from gmem cell's info */
#if 0
        const float4 cellSize = tex1Dfetch(texNodeSize,   cellIdx);
        const float4 cellPos  = tex1Dfetch(texNodeCenter, cellIdx);

#if 1
        const float4 cellCOM  = tex1Dfetch(texMultipole,  cellIdx+cellIdx+cellIdx);

        /* check if cell opening condition is satisfied */
        const float4 cellCOM1 = make_float4(cellCOM.x, cellCOM.y, cellCOM.z, cellPos.w);
        const bool splitCell = split_node_grav_impbh(cellCOM1, groupPos, groupSize);
#else /*added by egaburov, see compute_propertiesD.cu for matching code */
        const bool splitCell = split_node_grav_impbh(cellPos, groupPos, groupSize);
#endif
#else
        const float4   cellSize = tex1Dfetch(texCellSize, cellIdx);
        const CellData cellData = tex1Dfetch(texCellData, cellIdx);

        const bool splitCell = split_node_grav_impbh(cellSize, groupPos, groupSize);
#endif

        /* compute first child, either a cell if node or a particle if leaf */
#if 0
        const int cellData = __float_as_int(cellSize.w);
        const int firstChild =  cellData & 0x0FFFFFFF;
        const int nChildren  = (cellData & 0xF0000000) >> 28;
#endif

        /**********************************************/
        /* split cells that satisfy opening condition */
        /**********************************************/

        const bool isNode = cellData.isNode();

        {
          const int firstChild = cellData.first();
          const int nChildren  = cellData.n();
          bool splitNode  = isNode && splitCell && useCell;

          /* use exclusive scan to compute scatter addresses for each of the child cells */
          const int2 childScatter = warpIntExclusiveScan(nChildren & (-splitNode));

          /* make sure we still have available stack space */
          if (childScatter.y + nCells - cellListBlock > (CELL_LIST_MEM_PER_WARP<<SHIFT))
            return make_uint2(0xFFFFFFFF,0xFFFFFFFF);

#if 1
          /* if so populate next level stack in gmem */
          if (splitNode)
          {
            const int scatterIdx = cellListOffset + nCells + nextLevelCellCounter + childScatter.x;
            for (int i = 0; i < nChildren; i++)
              cellList[ringAddr<SHIFT>(scatterIdx + i)] = firstChild + i;
          }
#else  /* use scan operation to accomplish steps above, doesn't bring performance benefit */
          int nChildren  = childScatter.y;
          int nProcessed = 0;
          int2 scanVal   = {0};
          const int offset = cellListOffset + nCells + nextLevelCellCounter;
          while (nChildren > 0)
          {
            tmpList[laneIdx] = 1;
            if (splitNode && (childScatter.x - nProcessed < WARP_SIZE))
            {
              splitNode = false;
              tmpList[childScatter.x - nProcessed] = -1-firstChild;
            }
            scanVal = inclusive_segscan_warp(tmpList[laneIdx], scanVal.y);
            if (laneIdx < nChildren)
              cellList[ringAddr<SHIFT>(offset + nProcessed + laneIdx)] = scanVal.x;
            nChildren  -= WARP_SIZE;
            nProcessed += WARP_SIZE;
          }
#endif
          nextLevelCellCounter += childScatter.y;  /* increment nextLevelCounter by total # of children */
        }

#if 1
        {
          /***********************************/
          /******       APPROX          ******/
          /***********************************/

          /* see which thread's cell can be used for approximate force calculation */
          const bool approxCell    = !splitCell && useCell;
          const int2 approxScatter = warpBinExclusiveScan(approxCell);

          /* store index of the cell */
          const int scatterIdx = approxCounter + approxScatter.x;
          tmpList[laneIdx] = approxCellIdx;
          if (approxCell && scatterIdx < WARP_SIZE)
            tmpList[scatterIdx] = cellIdx;

          approxCounter += approxScatter.y;

          /* compute approximate forces */
          if (approxCounter >= WARP_SIZE)
          {
            /* evalute cells stored in shmem */
            approxAcc<NI,true>(acc_i, pos_i, tmpList[laneIdx], eps2);

            approxCounter -= WARP_SIZE;
            const int scatterIdx = approxCounter + approxScatter.x - approxScatter.y;
            if (approxCell && scatterIdx >= 0)
              tmpList[scatterIdx] = cellIdx;
            if (INTCOUNT)
              interactionCounters.x += WARP_SIZE*NI;
          }
          approxCellIdx = tmpList[laneIdx];
        }
#endif

#if 1
        {
          /***********************************/
          /******       DIRECT          ******/
          /***********************************/

          const bool isLeaf = !isNode;
          bool isDirect = splitCell && isLeaf && useCell;

          const int firstBody = cellData.pbeg();
          const int     nBody = cellData.pend() - cellData.pbeg();

          const int2 childScatter = warpIntExclusiveScan(nBody & (-isDirect));
          int nParticle  = childScatter.y;
          int nProcessed = 0;
          int2 scanVal   = {0};

          /* conduct segmented scan for all leaves that need to be expanded */
          while (nParticle > 0)
          {
            tmpList[laneIdx] = 1;
            if (isDirect && (childScatter.x - nProcessed < WARP_SIZE))
            {
              isDirect = false;
              tmpList[childScatter.x - nProcessed] = -1-firstBody;
            }
            scanVal = inclusive_segscan_warp(tmpList[laneIdx], scanVal.y);
            const int  ptclIdx = scanVal.x;

            if (nParticle >= WARP_SIZE)
            {
              directAcc<NI,true>(acc_i, pos_i, ptclIdx, eps2);
              nParticle  -= WARP_SIZE;
              nProcessed += WARP_SIZE;
              if (INTCOUNT)
                interactionCounters.y += WARP_SIZE*NI;
            }
            else 
            {
              const int scatterIdx = directCounter + laneIdx;
              tmpList[laneIdx] = directPtclIdx;
              if (scatterIdx < WARP_SIZE)
                tmpList[scatterIdx] = ptclIdx;

              directCounter += nParticle;

              if (directCounter >= WARP_SIZE)
              {
                /* evalute cells stored in shmem */
                directAcc<NI,true>(acc_i, pos_i, tmpList[laneIdx], eps2);
                directCounter -= WARP_SIZE;
                const int scatterIdx = directCounter + laneIdx - nParticle;
                if (scatterIdx >= 0)
                  tmpList[scatterIdx] = ptclIdx;
                if (INTCOUNT)
                  interactionCounters.y += WARP_SIZE*NI;
              }
              directPtclIdx = tmpList[laneIdx];

              nParticle = 0;
            }
          }
        }
#endif

        /* if the current level is processed, schedule the next level */
        if (cellListBlock >= nCells)
        {
          cellListOffset += nCells;
          nCells = nextLevelCellCounter;
          cellListBlock = nextLevelCellCounter = 0;
        }

      }  /* level completed */
#endif

#if 1
      if (approxCounter > 0)
      {
        approxAcc<NI,false>(acc_i, pos_i, laneIdx < approxCounter ? approxCellIdx : -1, eps2);
        if (INTCOUNT)
          interactionCounters.x += WARP_SIZE*NI; //approxCounter * NI;
        approxCounter = 0;
      }
#endif

#if 1
      if (directCounter > 0)
      {
        directAcc<NI,false>(acc_i, pos_i, laneIdx < directCounter ? directPtclIdx : -1, eps2);
        if (INTCOUNT)
          interactionCounters.y += WARP_SIZE*NI; //directCounter * NI;
        directCounter = 0;
      }
#endif

      return interactionCounters;
    }
}
#endif

  template<typename real_t, int NLEAF>
void Treecode<real_t, NLEAF>::computeForces()
{
  printf("Computing forces\n");
}

#include "TreecodeInstances.h"

